#include "hip/hip_runtime.h"
#include <string>

#include "caffe/util/gpu_math_functions.cuh"
#include "caffe/util/math_functions.hpp"

#pragma clang diagnostic push
#pragma ide diagnostic ignored "CannotResolve"
namespace caffe {

template<typename Dtype>
__global__ void LipschitzRegUpdateAllAndClear(int N,
		Dtype* g, Dtype *w,
		Dtype* h_g, Dtype* h_w,
    float rate,  float decay,
    bool reg_L2, bool clear_grads) {
  CUDA_KERNEL_LOOP(i, N) {
    h_g[i] = g[i];
    h_w[i] = w[i];
    float reg = reg_L2 ? (float)w[i] : float((Dtype(0.F) < w[i]) - (w[i] < Dtype(0.F)));
    float gr = float(g[i]) + reg * decay;
    w[i] -=  (Dtype) (gr * rate);
    g[i] = clear_grads ? Dtype(0) : g[i];
  }
}

#pragma clang diagnostic pop

/*
template<>
__global__ void LipschitzRegUpdateAllAndClear<half, half>(int N,
  half* g, half *w, half* m, half* v,
    float beta1, float beta2, float eps_hat, float local_rate, float local_decay,
    bool reg_L2,  bool clear_grads) {
  half hz;
  CUDA_KERNEL_LOOP(i, N) {
    float wf = __half2float(w[i]);
    float gf = __half2float(g[i]);
    float mf = __half2float(m[i]);
    float vf = __half2float(v[i]);

    float reg = reg_L2 ? wf : float((0.F < wf)-(wf < 0.F));
    gf += reg * local_decay;
    mf = beta1 * mf + (1.F - beta1)*gf;
    vf = beta2 * vf + (1.F - beta2)*gf*gf;
    gf = local_rate * mf / sqrt(vf + eps_hat);
    wf -= gf;

    w[i] = float2half_clip(wf);
    m[i] = float2half_clip(mf);
    v[i] = float2half_clip(vf);
    g[i] = clear_grads ? hz : float2half_clip(gf);
  }
}
*/

template<typename Dtype>
void Lipschitz_reg_update_and_clear_gpu(int N,
  Dtype* g,    Dtype *w,  Dtype* h_g,  Dtype* h_w,
  float rate, const std::string& reg_type, float decay,
  void *handle, bool clear_grads) {
  hipblasHandle_t cublas_handle =
      handle == nullptr ? Caffe::cublas_handle(0) : reinterpret_cast<hipblasHandle_t>(handle);
  hipStream_t stream;
  CUBLAS_CHECK(hipblasGetStream(cublas_handle, &stream));
  LipschitzRegUpdateAllAndClear  // NOLINT_NEXT_LINE(whitespace/operators)
      <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0, stream>>>(N,
      g, w, h_g, h_w, rate, decay, reg_type == "L2",
      clear_grads);
  CUDA_POST_KERNEL_CHECK;
  CUDA_CHECK(hipStreamSynchronize(stream));
}

template void Lipschitz_reg_update_and_clear_gpu<float16>(int, float16*, float16*, float16*, float16*, float,
    const std::string&, float, void*, bool);
template void Lipschitz_reg_update_and_clear_gpu<float>(int,   float*,   float*, float*, float*, float,
    const std::string&, float, void*, bool);
template void Lipschitz_reg_update_and_clear_gpu<double>(int,  double*,  double*, double*, double*, float,
    const std::string&, float, void*, bool);

}  // namespace caffe
